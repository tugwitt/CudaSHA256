#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include <dirent.h>
#include <ctype.h>


__global__ void sha256_cuda(JOB * job) {

	SHA256_CTX ctx;
	sha256_init(&ctx);
	sha256_update(&ctx, job->data, job->size);
	sha256_final(&ctx, job->digest);
}


void pre_sha256() {
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}


JOB * JOB_init(BYTE * data, long size) {
	JOB * j;
	checkCudaErrors(hipMallocManaged(&j, sizeof(JOB)));
	checkCudaErrors(hipMallocManaged(&(j->data), size));
	j->data = data;
	j->size = size;
	for (int i = 0; i < 64; i++)
	{
		j->digest[i] = 0xff;
	}
	return j;
}

void run_sha(unsigned char test[], char* string) {

	JOB * job;
	BYTE * buffer = 0;
	unsigned long fsize = strlen((char*)test);

	checkCudaErrors(hipMallocManaged(&buffer, (fsize+1)*sizeof(char)));
	
	memcpy(buffer, test, fsize);  
	job = JOB_init(buffer, fsize);

	pre_sha256();

	int blockSize = 4;
	int numBlocks = (1 + blockSize - 1) / blockSize;
	sha256_cuda <<< numBlocks, blockSize >>> (job);

	hipDeviceSynchronize();
	memcpy(string, hash_to_string(job->digest), 65);
	hipDeviceReset();

}




int main() {

	unsigned char test[] = "test\n";
	unsigned char test1[] = "test1\n";
	unsigned char test2[] = "test2\n";
	unsigned char test3[] = "test3\n";

	char string[65];

	run_sha(test, string);
	run_sha(test1, string);
	run_sha(test2, string);
	run_sha(test3, string);

	return 0;
}
