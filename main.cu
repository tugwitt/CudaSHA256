#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include <dirent.h>
#include <ctype.h>

char * trim(char *str){
    size_t len = 0;
    char *frontp = str;
    char *endp = NULL;

    if( str == NULL ) { return NULL; }
    if( str[0] == '\0' ) { return str; }

    len = strlen(str);
    endp = str + len;

    /* Move the front and back pointers to address the first non-whitespace
     * characters from each end.
     */
    while( isspace((unsigned char) *frontp) ) { ++frontp; }
    if( endp != frontp )
    {
        while( isspace((unsigned char) *(--endp)) && endp != frontp ) {}
    }

    if( str + len - 1 != endp )
            *(endp + 1) = '\0';
    else if( frontp != str &&  endp == frontp )
            *str = '\0';

    /* Shift the string so that it starts at str so that if it's dynamically
     * allocated, we can still free it on the returned pointer.  Note the reuse
     * of endp to mean the front of the string buffer now.
     */
    endp = str;
    if( frontp != str )
    {
            while( *frontp ) { *endp++ = *frontp++; }
            *endp = '\0';
    }


    return str;
}

__global__ void sha256_cuda(JOB ** jobs, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here


	if (i < n){
		SHA256_CTX ctx;
		sha256_init(&ctx);
		sha256_update(&ctx, jobs[i]->data, jobs[i]->size);
		sha256_final(&ctx, jobs[i]->digest);
	}
}


__global__ void sha256_cuda_new(const BYTE data[], size_t len, BYTE hash[]) {

	printf("%s", data);

	SHA256_CTX ctx;
	sha256_init(&ctx);
	sha256_update(&ctx, data, 5);
	sha256_final(&ctx, hash);
}


void pre_sha256() {
	// compy symbols
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}


void runJobs(JOB ** jobs, int n){
	int blockSize = 4;
	int numBlocks = (n + blockSize - 1) / blockSize;
	sha256_cuda <<< numBlocks, blockSize >>> (jobs, n);
}


JOB * JOB_init(BYTE * data, long size, char * fname) {
	JOB * j;
	checkCudaErrors(hipMallocManaged(&j, sizeof(JOB)));	//j = (JOB *)malloc(sizeof(JOB));
	checkCudaErrors(hipMallocManaged(&(j->data), size));
	j->data = data;
	j->size = size;
	for (int i = 0; i < 64; i++)
	{
		j->digest[i] = 0xff;
	}
	strcpy(j->fname, fname);
	return j;
}

int main(int argc, char **argv) {
	int i = 0, n = 0;
	unsigned long temp;
	BYTE * buffer = 0;
	char index;
	unsigned long fsize = 5;
	JOB ** jobs;

	checkCudaErrors(hipMallocManaged(&jobs, 1 * sizeof(JOB *)));
	checkCudaErrors(hipMallocManaged(&buffer, (fsize+1)*sizeof(char)));
	memcpy(buffer, "test\n", fsize);  

	jobs[0] = JOB_init(buffer, fsize, argv[index]);

	printf("%s", jobs[0]->data);

	pre_sha256();
	runJobs(jobs, 0);

	hipDeviceSynchronize();
	print_jobs(jobs, 0);
	hipDeviceReset();
	return 0;
}
